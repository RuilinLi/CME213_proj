#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_result));
    return result;
}

#define BLOCK_SIZE 16

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
// int myGEMM(double* __restrict__ A, double* __restrict__ B,
//            double* __restrict__ C, double* alpha, double* beta,
//            int M, int N, int K) {
//     /* TODO: Write an efficient GEMM implementation on GPU */
//     dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//     int num_block_x = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;
//     int num_block_y = (M + BLOCK_SIZE - 1)/BLOCK_SIZE;
//     dim3 blocks(num_block_x, num_block_y);

//     gpu_GEMM<<<blocks, threads>>>(A, B, C, *alpha, *beta, M, N, K);
//     return 0;
// }

// __global__
// void gpu_GEMM(const double* __restrict__ dA, const double* __restrict__ dB,
//               double* __restrict__ dC, double alpha, double beta,
//               int M, int N, int K)
// {
// // Note this implementation requires blockDim.y = blockDim.x
// // C is M by N, A is M by K, B is K by N
//     int Cx = blockIdx.x*blockDim.x + threadIdx.x;
//     int Cy = blockIdx.y*blockDim.y + threadIdx.y;

//     double C_val = 0.0;
//     int num_step = (K + BLOCK_SIZE - 1)/BLOCK_SIZE;
//     for(int i = 0; i < num_step; ++i){
//         __shared__ double As[BLOCK_SIZE * BLOCK_SIZE];
//         __shared__ double Bs[BLOCK_SIZE * BLOCK_SIZE];
//         int Ax_global = threadIdx.x + i*BLOCK_SIZE;
//         As[threadIdx.x*BLOCK_SIZE + threadIdx.y] = (Ax_global < K && Cy < M) ? dA[Ax_global*M + Cy]:0.0;
//         int By_global = threadIdx.y + i*BLOCK_SIZE;
//         Bs[threadIdx.x*BLOCK_SIZE + threadIdx.y] = (By_global < K && Cx < N) ? dB[Cx*K + By_global]:0.0;
//         __syncthreads();

//         for (int j = 0; j < BLOCK_SIZE; ++j){
//             C_val += As[threadIdx.y+BLOCK_SIZE*j]*Bs[j+BLOCK_SIZE*threadIdx.x];
//         }

//         __syncthreads();
//     }
//     if (Cx < N && Cy < M){
//         dC[Cx*M+Cy] = alpha*C_val + beta*dC[Cx*M+Cy];
//     }
// }

// Second implementation 
/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* __restrict__ A, double* __restrict__ B,
    double* __restrict__ C, double* alpha, double* beta,
    int M, int N, int K, hipStream_t stream) {
    /* TODO: Write an efficient GEMM implementation on GPU */
    dim3 threads(16, 4);
    int num_block_x = (N + 16 - 1)/16;
    int num_block_y = (M + 64 - 1)/64; // product of dimx and dimy
    dim3 blocks(num_block_x, num_block_y);
    gpu_GEMM<<<blocks, threads,0, stream>>>(A, B, C, *alpha, *beta, M, N, K);
    return 0;
}

// This is indeed faster
__global__ void 
gpu_GEMM(const double* __restrict__ dA, const double* __restrict__ dB,
               double* __restrict__ dC, double alpha, double beta,
               int M, int N, int K)
{

    constexpr int shared_x = 16; // same as blockdim.x
    constexpr int shared_y = 4; // same as a size and blockdim.y
    int sub_x =  shared_x;
    int sub_y = blockDim.x*blockDim.y;

    int col = blockIdx.x*sub_x;
    int row = blockIdx.y*sub_y;
    int Cx = blockIdx.x*sub_x + threadIdx.x;
    int num_step = (K + shared_y - 1)/shared_y;
    double a[shared_y];
    int row_offset = threadIdx.x+blockDim.x*threadIdx.y;
    double C_val[shared_x];
    for(int k = 0; k < shared_x; ++k){
        C_val[k] = 0.0;
    }
    __shared__ double Bs[shared_y*shared_x];
    for (int i  = 0; i < num_step; ++i){
        //Bs[threadIdx.x*shared_y+threadIdx.y] = (Cx < N && shared_y*i+threadIdx.y<K)?dB[Cx*K+shared_y*i+threadIdx.y]:0.0;
        // Make Bs row major
        Bs[threadIdx.x+threadIdx.y*shared_x] = (Cx < N && shared_y*i+threadIdx.y<K)?dB[Cx*K+shared_y*i+threadIdx.y]:0.0;

        for (int j = 0; j < shared_y; ++j){
            a[j] = (shared_y*i+j<K && row+row_offset<M)?dA[(shared_y*i+j)*M+row+row_offset]:0.0;
        }
        __syncthreads();
        // a[0] = (4*i<K && row+row_offset<M)?dA[(4*i)*M+row+row_offset]:0.0;
        // a[1] = (4*i+1<K && row+row_offset<M)?dA[(4*i+1)*M+row+row_offset]:0.0;
        // a[2] = (4*i+2<K && row+row_offset<M)?dA[(4*i+2)*M+row+row_offset]:0.0;
        // a[3] = (4*i+3<K && row+row_offset<M)?dA[(4*i+3)*M+row+row_offset]:0.0;

        for (int k = 0; k < shared_x; ++k){
            //C_val[k] += a[0]*Bs[k*4] + a[1]*Bs[k*4+1] +a[2]*Bs[k*4+2] +a[3]*Bs[k*4+3];
            for  (int j = 0; j< shared_y; ++j){
                //C_val[k] += a[j]*Bs[k*shared_y + j];
                // Make  Bs row major
                C_val[k] += a[j]*Bs[k+ j*shared_x];
            }
        }

        __syncthreads();
    }
    for (int k = 0; k < shared_x; ++k){
        if( col+k < N && row+row_offset <M){
            dC[(col+k)*M+row+row_offset] = alpha*C_val[k] + beta*dC[(col+k)*M+row+row_offset];
        }
    }
}

// This kernel add b to each column of Z in place
// Z is M by N, b has length M
__global__
void gpu_add_col(double* __restrict__ Z, const double* __restrict__ b, int M, int N){
    int indx = blockIdx.x*blockDim.x + threadIdx.x;
    int indy = blockIdx.y*blockDim.y + threadIdx.y;
    if (indx < N && indy < M){
        Z[indx*M+indy] += b[indy];
    }
}

int add_col(double* __restrict__ Z, const double* __restrict__ b, int M, int N, hipStream_t stream=0)
{
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    int num_block_x = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;
    int num_block_y = (M + BLOCK_SIZE - 1)/BLOCK_SIZE;
    dim3 blocks(num_block_x, num_block_y);

    gpu_add_col<<<blocks, threads, 0, stream>>>(Z, b, M, N);
    return 0;
}

// This kernel computes the sigmoid of the first matrix
// and save it to the second
// Z has dimension M by N
__global__
void sigmoid_gpu(const double* __restrict__ Z, double* __restrict__ a, int M, int N)
{
    int indx = blockIdx.x*blockDim.x + threadIdx.x;
    int indy = blockIdx.y*blockDim.y + threadIdx.y;
    if (indx < N && indy < M){
        a[indx*M+indy] = 1.0/(1+exp(-Z[indx*M+indy]));
    }

}

int sigmoid(const double* __restrict__ Z, double* __restrict__ a, int M, int N,hipStream_t stream=0)
{
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    int num_block_x = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;
    int num_block_y = (M + BLOCK_SIZE - 1)/BLOCK_SIZE;
    dim3 blocks(num_block_x, num_block_y);

    sigmoid_gpu<<<blocks, threads,0,stream>>>(Z, a, M, N);
    return 0;
}

// Function that computes softmax, assuming the output dimension
// is small
__global__
void softmax_gpu(const double* __restrict__ Z, double* __restrict__ a, int M, int N){
    int indx = blockIdx.x*blockDim.x + threadIdx.x;
    int indy = blockIdx.y*blockDim.y + threadIdx.y;
    if (indx < N && indy < M){
        double denom = 0.0;
        for (int i = 0; i < M; ++i){
            denom += exp(Z[M*indx + i]);
        }
        a[indx*M+indy] = exp(Z[M*indx+indy])/denom;
    }
}

int softmax(const double* __restrict__ Z, double* __restrict__ a, int M, int N,hipStream_t stream=0){
    dim3 threads(64, 2);
    int num_block_x = (N + 64 - 1)/64;
    int num_block_y = (M + 2 - 1)/2;
    dim3 blocks(num_block_x, num_block_y);
    softmax_gpu<<<blocks, threads,0,stream>>>(Z, a, M, N);
    return 0;
}


// Kernel that compute a * A + b * B and save it to C,
// a, b are scalars. A,B, C are M by N,
__global__
void matadd_gpu(const double* __restrict__ A, const double* __restrict__ B, double* __restrict__ C, int M, int N, double a, double b)
{    
    int indx = blockIdx.x*blockDim.x + threadIdx.x;
    int indy = blockIdx.y*blockDim.y + threadIdx.y;
    if (indx < N && indy < M){
        C[indx*M + indy] = a*A[indx*M + indy] + b*B[indx*M + indy];
    }
}

int matadd(const double* __restrict__ A, const double* __restrict__ B, double* __restrict__ C, int M, int N, double a, double b, hipStream_t stream=0)
{
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    int num_block_x = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;
    int num_block_y = (M + BLOCK_SIZE - 1)/BLOCK_SIZE;
    dim3 blocks(num_block_x, num_block_y);

    matadd_gpu<<<blocks, threads,0,stream>>>(A, B, C, M, N, a, b);
    return 0;

}


// This kernel transpose the matrix A and save it to
// At, A is M by N, for now do it naively
__global__
void transpose_gpu(const double* __restrict__ A, double* __restrict__ At, int M, int N){
    int indx = blockIdx.x*blockDim.x + threadIdx.x;
    int indy = blockIdx.y*blockDim.y + threadIdx.y;
    if(indx < N && indy < M){
        At[indx + indy*N] = A[indy + indx*M];
    }
}

int transpose(const double* __restrict__ A, double* __restrict__ At, int M, int N,hipStream_t stream=0){
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    int num_block_x = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;
    int num_block_y = (M + BLOCK_SIZE - 1)/BLOCK_SIZE;
    dim3 blocks(num_block_x, num_block_y);

    transpose_gpu<<<blocks, threads,0,stream>>>(A, At, M, N);
    return 0;
}

// This kernel sum the rows of the matrix A and store 
// it to the entries of b. A is M by N
__global__
void naive_reduce_sum(const double* __restrict__ A, double* __restrict__ b, int M, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < M){
        double result = 0.0;
        for (int j = 0; j < N; ++j){
            result += A[idx+j*M];
        }
        b[idx] = result;
    }
}

int naive_sum(const double* __restrict__ A, double* __restrict__ b, int M, int N,hipStream_t stream=0)
{
    int thread = 1;
    int block = M;
    naive_reduce_sum<<<block, thread,0,stream>>>(A, b, M, N);
    return 0;
}

// This is a specialized kernel to compute dCE/dz1
// dz1 has dimension M by N
__global__
void get_dz1_gpu(double* __restrict__ dz1, const double* __restrict__ a, int M, int N)
{
    int indx = blockIdx.x*blockDim.x + threadIdx.x;
    int indy = blockIdx.y*blockDim.y + threadIdx.y;
    if (indx < N && indy < M){
        dz1[indx*M+indy] *= (a[indx*M+indy]*(1-a[indx*M+indy]));
    }
}

int get_dz1(double* __restrict__ dz1, const double* __restrict__ a, int M, int N,hipStream_t stream=0){
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    int num_block_x = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;
    int num_block_y = (M + BLOCK_SIZE - 1)/BLOCK_SIZE;
    dim3 blocks(num_block_x, num_block_y);
    get_dz1_gpu<<<blocks, threads,0,stream>>>(dz1, a, M, N);
    return 0;
}


void allocate_device_memory(raw_params &d_params,
                            raw_cache &d_cache,
                            raw_grad &d_grad,
                            raw_bp &d_bp,
                            const std::vector<int>& H,
                            int batch_size_node)
{
    hipMalloc((void**)&d_params.W1, sizeof(double) * H[0] * H[1]);
    hipMalloc((void**)&d_params.b1, sizeof(double) * H[1]);
    hipMalloc((void**)&d_params.W2, sizeof(double) * H[1] * H[2]);
    hipMalloc((void**)&d_params.b2, sizeof(double) * H[2]);

    hipMalloc((void**)&d_cache.X, sizeof(double) * H[0] * batch_size_node);
    hipMalloc((void**)&d_cache.z1, sizeof(double) * H[1] * batch_size_node);
    hipMalloc((void**)&d_cache.a1, sizeof(double) * H[1] * batch_size_node);
    hipMalloc((void**)&d_cache.z2, sizeof(double) * H[2] * batch_size_node);
    hipMalloc((void**)&d_cache.y, sizeof(double) * H[2] * batch_size_node);
    hipMalloc((void**)&d_cache.yhat, sizeof(double) * H[2] * batch_size_node);

    hipMalloc((void**)&d_grad.dW1, sizeof(double) * H[0] * H[1]);
    hipMalloc((void**)&d_grad.db1, sizeof(double) * H[1]);
    hipMalloc((void**)&d_grad.dW2, sizeof(double) * H[1] * H[2]);
    hipMalloc((void**)&d_grad.db2, sizeof(double) * H[2]);

    hipMalloc((void**)&d_bp.ydiff, sizeof(double) * H[2]*batch_size_node);
    hipMalloc((void**)&d_bp.a1t, sizeof(double) * H[1] * batch_size_node);
    hipMalloc((void**)&d_bp.W2t, sizeof(double) * H[1] * H[2]);
    hipMalloc((void**)&d_bp.Xt, sizeof(double) * H[0] * batch_size_node);
    hipMalloc((void**)&d_bp.dz1, sizeof(double) * H[1] * batch_size_node);
    

}


// void send_data_to_device(const double *X_local, 
//                          const double *y_local, 
//                          raw_cache &d_cache, 
//                          int batch_size_node, 
//                          int input_dim, 
//                          int output_dim,
//                          hipStream_t mystream[])
// {
//     // hipMemcpyAsync(d_cache.X, X_local, sizeof(double) * batch_size_node * input_dim, hipMemcpyHostToDevice, mystream[3]);
//     // hipMemcpyAsync(d_cache.y, y_local, sizeof(double) * batch_size_node * output_dim, hipMemcpyHostToDevice, mystream[2]);
//     hipMemcpy(d_cache.X, X_local, sizeof(double) * batch_size_node * input_dim, hipMemcpyHostToDevice);
//     hipMemcpy(d_cache.y, y_local, sizeof(double) * batch_size_node * output_dim, hipMemcpyHostToDevice);
// }

void send_data_to_device(const double *src, 
    double *dest, 
    int batch_size_node, 
    int dim,
    hipStream_t stream)
{
    hipMemcpyAsync(dest, src, sizeof(double) * batch_size_node * dim, hipMemcpyHostToDevice, stream);
// hipMemcpy(dest, X_local, sizeof(double) * batch_size_node * input_dim, hipMemcpyHostToDevice);
// hipMemcpy(d_cache.y, y_local, sizeof(double) * batch_size_node * output_dim, hipMemcpyHostToDevice);
}


// size is the number of images in this batch
void forward_pass(raw_params &d_params, raw_cache &d_cache, int input_dim, int h1, int output_dim, int size, hipStream_t mystream[])
{
    double alpha = 1.0;
    double beta = 0.0;
    myGEMM(d_params.W1, d_cache.X, d_cache.z1, &alpha, &beta, h1, size, input_dim, mystream[0]);
    add_col(d_cache.z1, d_params.b1, h1, size, mystream[0]);
    sigmoid(d_cache.z1, d_cache.a1, h1, size, mystream[0]);
    myGEMM(d_params.W2, d_cache.a1, d_cache.z2, &alpha, &beta, output_dim, size, h1, mystream[0]);
    add_col(d_cache.z2, d_params.b2, output_dim, size, mystream[0]);
    softmax(d_cache.z2, d_cache.yhat, output_dim, size, mystream[0]);
}

void backward_pass(raw_params &d_params,
                   raw_cache &d_cache,
                   raw_grad &d_grad, 
                   raw_bp &d_bp,
                   int input_dim, 
                   int h1, 
                   int output_dim,
                   double reg,
                   int size,
                   int batch_size,
                   int num_procs,
                   hipStream_t mystream[])
{
    double alpha = 1.0;
    double beta = 0.0;
    hipEvent_t event; 
    hipEventCreate (&event);

    matadd(d_cache.yhat, d_cache.y, d_bp.ydiff, output_dim, size, 1.0/(double)batch_size, -1.0/(double)batch_size, mystream[0]);
    transpose(d_cache.a1, d_bp.a1t, h1, size, mystream[1]);
    transpose(d_params.W2, d_bp.W2t, output_dim, h1, mystream[2]);
    transpose(d_cache.X, d_bp.Xt, input_dim, size, mystream[3]);
    hipStreamSynchronize(mystream[0]);
    myGEMM(d_bp.ydiff, d_bp.a1t, d_grad.dW2, &alpha, &beta, output_dim, h1, size, mystream[1]);
    matadd(d_grad.dW2, d_params.W2, d_grad.dW2, output_dim, h1, 1.0, reg/(double)num_procs, mystream[1]);

    naive_sum(d_bp.ydiff, d_grad.db2, output_dim, size, mystream[3]);
    // compute partial w.r.t a1
    myGEMM(d_bp.W2t, d_bp.ydiff, d_bp.dz1, &alpha, &beta, h1, size ,output_dim, mystream[0]);
    get_dz1(d_bp.dz1, d_cache.a1, h1, size, mystream[0]);
    hipEventRecord(event, mystream[0]);
    myGEMM(d_bp.dz1, d_bp.Xt, d_grad.dW1, &alpha, &beta, h1, input_dim, size, mystream[0]);

    hipStreamWaitEvent(mystream[2], event, 0);
    naive_sum(d_bp.dz1, d_grad.db1, h1, size, mystream[2]);

    // Add regularization terms to the grads
    matadd(d_grad.dW1, d_params.W1, d_grad.dW1, h1, input_dim, 1.0, reg/(double)num_procs, mystream[0]);
}

void gradient_descent(raw_grad &d_grad, 
                      raw_params &d_params, 
                      double learning_rate,
                      int input_dim,
                      int h1,
                      int output_dim,
                      hipStream_t mystream[])
{
    matadd(d_params.W1, d_grad.dW1, d_params.W1, h1, input_dim, 1.0, -learning_rate, mystream[0]);
    matadd(d_params.W2, d_grad.dW2, d_params.W2, output_dim, h1, 1.0, -learning_rate, mystream[1]);
    matadd(d_params.b1, d_grad.db1, d_params.b1, h1, 1, 1.0, -learning_rate, mystream[2]);
    matadd(d_params.b2, d_grad.db2, d_params.b2, output_dim, 1, 1.0, -learning_rate, mystream[3]);
}




void free_all_CUDA(raw_params &d_params, raw_cache &d_cache, raw_grad &d_grad)
{
    hipFree(d_params.W1);
    hipFree(d_params.W2);
    hipFree(d_params.b1);
    hipFree(d_params.b2);
    hipFree(d_cache.X);
    hipFree(d_cache.z1);
    hipFree(d_cache.a1);
    hipFree(d_cache.z2);
    hipFree(d_cache.y);
    hipFree(d_cache.yhat);
    hipFree(d_grad.dW1);
    hipFree(d_grad.dW2);
    hipFree(d_grad.db1);
    hipFree(d_grad.db2);
}

